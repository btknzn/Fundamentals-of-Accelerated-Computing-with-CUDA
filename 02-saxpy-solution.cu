
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nvprof to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c, int stride)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tid < N; tid+=stride)
    {
        c[tid] = a[tid] * 2 + b[tid];
    }
}
__global__ void init(int *a, int val, int stride)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (; tid < N; tid+=stride)
    {
        a[tid] = val;
    }
}
int main()
{
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    
    int deviceId = 0;
    hipGetDevice(&deviceId);
    int SMc = 1;
    hipDeviceGetAttribute(&SMc, hipDeviceAttributeMultiprocessorCount, deviceId);
    int threads_per_block = 256;
    int number_of_blocks = 32 * SMc;
    int stride = threads_per_block * number_of_blocks;
    init<<<number_of_blocks, threads_per_block>>>(a,2,stride);
    init<<<number_of_blocks, threads_per_block>>>(b,1,stride);
    init<<<number_of_blocks, threads_per_block>>>(c,0,stride);
    hipDeviceSynchronize();
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c, stride );
    hipDeviceSynchronize();

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
